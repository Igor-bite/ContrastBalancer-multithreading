#include "hip/hip_runtime.h"
//
// Created by Igor Kluzhev on 25.09.2024.
//

#include "pnm.h"
#include <cmath>
#include <stdio.h>
#include <stdexcept>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

using namespace std;

PNMPicture::PNMPicture() = default;
PNMPicture::PNMPicture(const string& filename) {
    read(filename);
}

PNMPicture::~PNMPicture() {
    if (fin != nullptr) {
        fclose(fin);
    }
    if (fout != nullptr) {
        fclose(fout);
    }
}

void PNMPicture::read(const string& fileName) {
    fin = fopen(fileName.c_str(), "rb");
    if (fin == nullptr) {
        throw runtime_error("Error while trying to open input file");
    }

    char p;
    char binChar;
    fscanf(fin, "%c%i%c%d %d%c%d%c", &p, &format, &binChar, &width, &height, &binChar, &colors, &binChar);

    if (p != 'P')
        throw runtime_error("Unsupported format input file");

    read();
    fclose(fin);
    fin = nullptr;
}

void PNMPicture::read() {
    if (format == 5) {
        channelsCount = 1;
    } else if (format == 6) {
        channelsCount = 3;
    } else {
        throw runtime_error("Unsupported format of PNM file");
    }
    data_size = width * height * channelsCount;
    data.resize(data_size);

    const size_t bytesRead = fread(data.data(), 1, data_size, fin);

    if (bytesRead != data_size) {
        throw runtime_error("Error while trying to read file");
    }
}

void PNMPicture::write(const string& fileName) {
    fout = fopen(fileName.c_str(), "wb");
    if (fout == nullptr) {
        throw runtime_error("Error while trying to open output file");
    }

    write();
    fclose(fout);
    fout = nullptr;
}

void PNMPicture::write() {
    fprintf(fout, "P%d\n%d %d\n%d\n", format, width, height, colors);

    size_t dataSize = width * height * channelsCount;
    const size_t writtenBytes = fwrite(data.data(), 1, dataSize, fout);

    if (writtenBytes != dataSize) {
        throw runtime_error("Error while trying to write to file");
    }
}

// 1) изначально при итерировании собираем кол-ва по каждому цвету
// 2) при итерировании по цветам суммируем кол-во для тёмных и светлых
// 3) при достижении нужного кол-ва - идём дальше
// и сохраняем первый попавшийся индекс с ненулевым значением как минимальный/максимальный цвет
// 4) вычисляем min/max
// 5) пробегаемся ещё раз и меняем значения
// доступные методы: omp + simd + ilp

void PNMPicture::modifyParallelCUDA(const float coeff, const int device_index) noexcept {
    if (data_size == 1) {
        return;
    }

    size_t ignoreCount = data_size * coeff;
    vector<size_t> elements;
    uchar min_v = 255;
    uchar max_v = 0;

    analyzeDataParallelCUDA(elements);
    determineMinMax(ignoreCount, elements, min_v, max_v);

    // если уже растянуто - не делаем ничего
    // или если например 1 цвет - не делаем ничего
    if ((min_v == 0 && max_v == 255) || min_v >= max_v) {
        return;
    }

    float const scale = 255 / float(max_v - min_v);
    float scaledMinV = scale * float(min_v);

    uchar* d = data.data();
    for (size_t i = 0; i < data_size; i++) {
        int scaledValue1 = scale * d[i] - scaledMinV;
        d[i] = max(0, min(scaledValue1, 255));
    }
}

void PNMPicture::determineMinMax(
    size_t ignoreCount,
    const vector<size_t> &elements,
    uchar &min_v,
    uchar &max_v
) const noexcept {
    int darkCount = 0;
    bool isDarkComplete = false;

    int brightCount = 0;
    bool isBrightComplete = false;

    for (size_t i = 0; i < 256; i++) {
        if (!isDarkComplete) {
            size_t darkIndex = i;
            int element = elements[darkIndex];
            if (darkCount < ignoreCount) {
                darkCount += element;
            }

            if (darkCount >= ignoreCount && element != 0) {
                isDarkComplete = true;
                min_v = darkIndex;
            }
        }
        if (!isBrightComplete) {
            size_t brightIndex = 255 - i;
            int element = elements[brightIndex];
            if (brightCount < ignoreCount) {
                brightCount += element;
            }

            if (brightCount >= ignoreCount && element != 0) {
                isBrightComplete = true;
                max_v = brightIndex;
            }
        }

        if (isDarkComplete && isBrightComplete) {
            break;
        }
    }

    for (size_t i = 0; i < 256; i++) {
        size_t brightIndex = 255 - i;
        int element = elements[brightIndex];
        if (brightCount < ignoreCount) {
            brightCount += element;
        }

        if (brightCount >= ignoreCount && element != 0) {
            max_v = brightIndex;
            break;
        }
    }
}

void PNMPicture::analyzeDataParallelCUDA(
    vector<size_t> &elements
) const noexcept {
    elements.resize(256, 0);

    const uchar* d = data.data();
    for (size_t i = 0; i < data_size; i++) {
        elements[d[i]] += 1;
    }
}