#include <hip/hip_runtime.h>

#include <stdio.h>
using namespace std;

// Cheat sheet -> https://kdm.icm.edu.pl/Tutorials/GPU-intro/introduction.en/#:~:text=The%20maximum%20number%20of%20threads,a%20block%20of%201024%20threads.

#define EXIT_CODE_ERROR 1
#define CUDA_WRAP_ERROR(FUNCTION)                                   \
if (auto err = FUNCTION) {                                          \
    fprintf(stderr, "%s:%d function failed!", __FILE__, __LINE__);  \
    return EXIT_CODE_ERROR;                                         \
}

__global__ void kernel_add(int *c, const int *a, const int *b)
{
    unsigned int x = threadIdx.x;
    c[x] = a[x] + b[x];
}

void printArray(int* a, int size)
{
    printf("{");
    for (size_t i = 0; i < size - 1; i++)
    {
        printf("%i, ", a[i]);
    }
    if (size > 0)
    {
        printf("%i", a[size - 1]);
    }
    printf("}");
}

int main()
{
    int i;

    CUDA_WRAP_ERROR(hipGetDeviceCount(&i));
    fprintf(stdout, "Device count: %i\n", i);

    hipDeviceProp_t prop{};
    CUDA_WRAP_ERROR(hipGetDeviceProperties(&prop, 0));

    fprintf(stdout, "Name: %s\n", prop.name);
    fprintf(stdout, "major.minor: %i.%i\n\n", prop.major, prop.minor);

    CUDA_WRAP_ERROR(hipSetDevice(0));

    constexpr int array_size = 5;
    constexpr int byte_size = array_size * sizeof(int);
    int a[array_size] = { 1, 2, 3, 4, 5 };
    int b[array_size] = { 10, 20, 30, 40, 50 };
    int c[array_size];

    int *dev_a, *dev_b, *dev_c;

    CUDA_WRAP_ERROR(hipMalloc(&dev_a, byte_size));
    CUDA_WRAP_ERROR(hipMalloc(&dev_b, byte_size));
    CUDA_WRAP_ERROR(hipMalloc(&dev_c, byte_size));

    CUDA_WRAP_ERROR(hipMemcpyAsync(dev_a, a, byte_size, hipMemcpyKind::hipMemcpyHostToDevice));
    CUDA_WRAP_ERROR(hipMemcpyAsync(dev_b, b, byte_size, hipMemcpyKind::hipMemcpyHostToDevice));

    kernel_add<<<1, 5>>>(dev_c, dev_a, dev_b);
    CUDA_WRAP_ERROR(hipGetLastError());

    CUDA_WRAP_ERROR(hipMemcpy(c, dev_c, byte_size, hipMemcpyKind::hipMemcpyDeviceToHost));

    printArray(a, array_size);
    printf(" + ");
    printArray(b, array_size);
    printf(" = ");
    printArray(c, array_size);
    printf("\n\n");

    CUDA_WRAP_ERROR(hipFree(dev_a));
    CUDA_WRAP_ERROR(hipFree(dev_b));
    CUDA_WRAP_ERROR(hipFree(dev_c));

    return 0;
}
